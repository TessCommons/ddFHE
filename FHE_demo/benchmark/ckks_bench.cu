#include "hip/hip_runtime.h"
#include "bench_utils.cuh"
#include "util.cuh"

using namespace std;
using namespace phantom;
using namespace phantom::arith;
using namespace phantom::util;

void ckks_performance_test(EncryptionParameters &parms, double scale) {
    PhantomContext context(parms);
    print_parameters(context);
    cout << endl;

    cuda_stream_wrapper stream;

    print_timer_banner();

    auto count = 100;

    {
        CUDATimer timer("gen_secretkey");
        for (auto i = 0; i < count; i++) {
            timer.start();
            PhantomSecretKey secret_key(context);
            timer.stop();
        }
    }

    PhantomSecretKey secret_key(context);

    {
        CUDATimer timer("gen_publickey");
        for (auto i = 0; i < count; i++) {
            timer.start();
            PhantomPublicKey public_key = secret_key.gen_publickey(context);
            timer.stop();
        }
    }

    PhantomPublicKey public_key = secret_key.gen_publickey(context);

    // Generate relinearization keys
    {
        CUDATimer timer("gen_relinkey");
        for (auto i = 0; i < count; i++) {
            timer.start();
            PhantomRelinKey relin_keys = secret_key.gen_relinkey(context);
            timer.stop();
        }
    }

    PhantomRelinKey relin_keys = secret_key.gen_relinkey(context);

    PhantomGaloisKey gal_keys = secret_key.create_galois_keys(context);

    PhantomCKKSEncoder ckks_encoder(context);

    /*
    Populate a vector of floating-point values to batch.
    */
    std::vector<hipDoubleComplex> x;
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(0.0, 1.0);
    for (size_t i = 0; i < ckks_encoder.slot_count(); i++) {
        x.push_back(make_hipDoubleComplex(dis(gen), dis(gen)));
    }

    /*
    [Encoding]
    For scale we use the square root of the last coeff_modulus prime
    from parms.
    */
    PhantomPlaintext plain;
    {
        CUDATimer timer("encode");
        for (auto i = 0; i < count; i++) {
            timer.start();
            ckks_encoder.encode(context, x, scale, plain, 1);
            timer.stop();
        }
    }

    /*
    [Decoding]
    */
    {
        CUDATimer timer("decode");
        for (auto i = 0; i < count; i++) {
            timer.start();
            auto pod_vector2 = ckks_encoder.decode<hipDoubleComplex>(context, plain);
            timer.stop();
        }
    }

    /*
    [Encryption]
    */
    PhantomCiphertext encrypted;
    {
        CUDATimer timer("encrypt_asymmetric");
        for (auto i = 0; i < count; i++) {
            timer.start();
            public_key.encrypt_asymmetric(context, plain, encrypted);
            timer.stop();
        }
    }

    /*
    [Decryption]
    */
    PhantomPlaintext plain2;
    {
        CUDATimer timer("decrypt");
        for (auto i = 0; i < count; i++) {
            timer.start();
            secret_key.decrypt(context, encrypted, plain2);
            timer.stop();
        }
    }

    // homomorphic operations
    std::vector<hipDoubleComplex> pod_vector3(ckks_encoder.slot_count());
    std::vector<hipDoubleComplex> pod_vector4(ckks_encoder.slot_count());

    PhantomCiphertext encrypted1;
    for (size_t j = 0; j < ckks_encoder.slot_count(); j++)
        pod_vector3[j] = make_hipDoubleComplex(double(1), double(0));
    ckks_encoder.encode(context, pod_vector3, scale, plain, 1);
    public_key.encrypt_asymmetric(context, plain, encrypted1);

    PhantomCiphertext encrypted2;
    for (size_t j = 0; j < ckks_encoder.slot_count(); j++)
        pod_vector4[j] = make_hipDoubleComplex(double(1), double(0));
    ckks_encoder.encode(context, pod_vector4, scale, plain2, 1);
    public_key.encrypt_asymmetric(context, plain2, encrypted2);

    /*
    [Add]
    */
    {
        CUDATimer timer("add");
        for (auto i = 0; i < count; i++) {
            PhantomCiphertext tmp_ct(encrypted1);
            timer.start();
            add_inplace(context, tmp_ct, encrypted2);
            timer.stop();
        }
    }

    /*
    [Add Plain]
    */
    {
        CUDATimer timer("add_plain");
        for (auto i = 0; i < count; i++) {
            PhantomCiphertext tmp_ct(encrypted1);
            timer.start();
            add_plain_inplace(context, tmp_ct, plain);
            timer.stop();
        }
    }

    /*
    [Multiply]
    */
    {
        CUDATimer timer("multiply");
        for (auto i = 0; i < count; i++) {
            PhantomCiphertext tmp_ct(encrypted1);
            timer.start();
            multiply_inplace(context, tmp_ct, encrypted2);
            relinearize_inplace(context, tmp_ct, relin_keys);
            timer.stop();
        }
    }

    /*
    [Multiply Plain]
    */
    {
        CUDATimer timer("multiply_plain");
        for (auto i = 0; i < count; i++) {
            PhantomCiphertext tmp_ct(encrypted1);
            timer.start();
            multiply_plain_inplace(context, tmp_ct, plain);
            timer.stop();
        }
    }

    /*
    [Rescale]
    */
    {
        CUDATimer timer("rescale_to_next");
        for (auto i = 0; i < count; i++) {
            PhantomCiphertext tmp_ct(encrypted1);
            multiply_inplace(context, tmp_ct, encrypted2);
            relinearize_inplace(context, tmp_ct, relin_keys);
            timer.start();
            rescale_to_next_inplace(context, tmp_ct);
            timer.stop();
        }
    }

    /*
    [Rotate Vector]
    */
    {
        CUDATimer timer("rotate_vector_one_step");
        for (auto i = 0; i < count; i++) {
            PhantomCiphertext tmp_ct(encrypted1);
            timer.start();
            rotate_inplace(context, tmp_ct, 1, gal_keys);
            timer.stop();
        }
    }
}

int main() {
    print_example_banner("CKKS Performance Test with Degrees: 4096, 8192, 16384, 32768, and 65536");

    std::vector<int> galois_steps = {1};

    // 2 ^ 13

    {
        EncryptionParameters parms(scheme_type::ckks);
        size_t poly_modulus_degree = 1 << 13;
        parms.set_poly_modulus_degree(poly_modulus_degree);
        parms.set_galois_elts(get_elts_from_steps(galois_steps, poly_modulus_degree));
        parms.set_coeff_modulus(CoeffModulus::Create(poly_modulus_degree, {60, 40, 60}));
        parms.set_special_modulus_size(1);
        double scale = pow(2.0, 40);
        ckks_performance_test(parms, scale);
    }
    {
        EncryptionParameters parms(scheme_type::ckks);
        size_t poly_modulus_degree = 1 << 13;
        parms.set_poly_modulus_degree(poly_modulus_degree);
        parms.set_galois_elts(get_elts_from_steps(galois_steps, poly_modulus_degree));
        parms.set_coeff_modulus(CoeffModulus::Create(poly_modulus_degree, {60, 40, 40, 60}));
        parms.set_special_modulus_size(1);
        double scale = pow(2.0, 40);
        ckks_performance_test(parms, scale);
    }

    // 2 ^ 14

    {
        EncryptionParameters parms(scheme_type::ckks);
        size_t poly_modulus_degree = 1 << 14;
        parms.set_poly_modulus_degree(poly_modulus_degree);
        parms.set_galois_elts(get_elts_from_steps(galois_steps, poly_modulus_degree));
        parms.set_coeff_modulus(CoeffModulus::Create(poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 60}));
        parms.set_special_modulus_size(1);
        double scale = pow(2.0, 40);
        ckks_performance_test(parms, scale);
    }
    {
        EncryptionParameters parms(scheme_type::ckks);
        size_t poly_modulus_degree = 1 << 14;
        parms.set_poly_modulus_degree(poly_modulus_degree);
        parms.set_galois_elts(get_elts_from_steps(galois_steps, poly_modulus_degree));
        parms.set_coeff_modulus(CoeffModulus::Create(poly_modulus_degree, {60, 40, 40, 40, 40, 40, 60, 60}));
        parms.set_special_modulus_size(2);
        double scale = pow(2.0, 40);
        ckks_performance_test(parms, scale);
    }

    // 2 ^ 15

    {
        EncryptionParameters parms(scheme_type::ckks);
        size_t poly_modulus_degree = 1 << 15;
        parms.set_poly_modulus_degree(poly_modulus_degree);
        parms.set_galois_elts(get_elts_from_steps(galois_steps, poly_modulus_degree));
        parms.set_coeff_modulus(CoeffModulus::Create(poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                                                                           40, 40, 40, 40, 40, 40, 40, 40, 40, 60}));
        parms.set_special_modulus_size(1);
        double scale = pow(2.0, 40);
        ckks_performance_test(parms, scale);
    }
    {
        EncryptionParameters parms(scheme_type::ckks);
        size_t poly_modulus_degree = 1 << 15;
        parms.set_poly_modulus_degree(poly_modulus_degree);
        parms.set_galois_elts(get_elts_from_steps(galois_steps, poly_modulus_degree));
        parms.set_coeff_modulus(CoeffModulus::Create(
                poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 60, 60}));
        parms.set_special_modulus_size(2);
        double scale = pow(2.0, 40);
        ckks_performance_test(parms, scale);
    }
    {
        EncryptionParameters parms(scheme_type::ckks);
        size_t poly_modulus_degree = 1 << 15;
        parms.set_poly_modulus_degree(poly_modulus_degree);
        parms.set_galois_elts(get_elts_from_steps(galois_steps, poly_modulus_degree));
        parms.set_coeff_modulus(CoeffModulus::Create(
                poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 60, 60, 60}));
        parms.set_special_modulus_size(3);
        double scale = pow(2.0, 40);
        ckks_performance_test(parms, scale);
    }
    {
        EncryptionParameters parms(scheme_type::ckks);
        size_t poly_modulus_degree = 1 << 15;
        parms.set_poly_modulus_degree(poly_modulus_degree);
        parms.set_galois_elts(get_elts_from_steps(galois_steps, poly_modulus_degree));
        parms.set_coeff_modulus(CoeffModulus::Create(poly_modulus_degree,
                                                     {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 60, 60, 60, 60}));
        parms.set_special_modulus_size(4);
        double scale = pow(2.0, 40);
        ckks_performance_test(parms, scale);
    }

    // 2^ 16

    {
        EncryptionParameters parms(scheme_type::ckks);
        size_t poly_modulus_degree = 1 << 16;
        parms.set_poly_modulus_degree(poly_modulus_degree);
        parms.set_galois_elts(get_elts_from_steps(galois_steps, poly_modulus_degree));
        parms.set_coeff_modulus(
                CoeffModulus::Create(poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                                                           40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                                                           40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 60}));
        parms.set_special_modulus_size(1);
        double scale = pow(2.0, 40);
        ckks_performance_test(parms, scale);
    }
    {
        EncryptionParameters parms(scheme_type::ckks);
        size_t poly_modulus_degree = 1 << 16;
        parms.set_poly_modulus_degree(poly_modulus_degree);
        parms.set_galois_elts(get_elts_from_steps(galois_steps, poly_modulus_degree));
        parms.set_coeff_modulus(
                CoeffModulus::Create(poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                                                           40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                                                           40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 60, 60}));
        parms.set_special_modulus_size(2);
        double scale = pow(2.0, 40);
        ckks_performance_test(parms, scale);
    }
    {
        EncryptionParameters parms(scheme_type::ckks);
        size_t poly_modulus_degree = 1 << 16;
        parms.set_poly_modulus_degree(poly_modulus_degree);
        parms.set_galois_elts(get_elts_from_steps(galois_steps, poly_modulus_degree));
        parms.set_coeff_modulus(
                CoeffModulus::Create(poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                                                           40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                                                           40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 60, 60, 60}));
        parms.set_special_modulus_size(3);
        double scale = pow(2.0, 40);
        ckks_performance_test(parms, scale);
    }
    {
        EncryptionParameters parms(scheme_type::ckks);
        size_t poly_modulus_degree = 1 << 16;
        parms.set_poly_modulus_degree(poly_modulus_degree);
        parms.set_galois_elts(get_elts_from_steps(galois_steps, poly_modulus_degree));
        parms.set_coeff_modulus(CoeffModulus::Create(
                poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                                      40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 60, 60, 60, 60}));
        parms.set_special_modulus_size(4);
        double scale = pow(2.0, 40);
        ckks_performance_test(parms, scale);
    }
    {
        EncryptionParameters parms(scheme_type::ckks);
        size_t poly_modulus_degree = 1 << 16;
        parms.set_poly_modulus_degree(poly_modulus_degree);
        parms.set_galois_elts(get_elts_from_steps(galois_steps, poly_modulus_degree));
        parms.set_coeff_modulus(CoeffModulus::Create(
                poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                                      40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 60, 60, 60, 60, 60}));
        parms.set_special_modulus_size(5);
        double scale = pow(2.0, 40);
        ckks_performance_test(parms, scale);
    }
    {
        EncryptionParameters parms(scheme_type::ckks);
        size_t poly_modulus_degree = 1 << 16;
        parms.set_poly_modulus_degree(poly_modulus_degree);
        parms.set_galois_elts(get_elts_from_steps(galois_steps, poly_modulus_degree));
        parms.set_coeff_modulus(CoeffModulus::Create(
                poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                                      40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 60, 60, 60, 60, 60, 60}));
        parms.set_special_modulus_size(6);
        double scale = pow(2.0, 40);
        ckks_performance_test(parms, scale);
    }

    return 0;
}
