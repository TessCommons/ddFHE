#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "cuda_wrapper.cuh"

using namespace phantom::util;

#define EPSINON 0.001

inline bool operator==(const hipDoubleComplex &lhs, const hipDoubleComplex &rhs) {
    return fabs(lhs.x - rhs.x) < EPSINON;
}

__global__ void scaling_kernel(hipDoubleComplex *data, int element_count, float scale) {
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const int stride = blockDim.x * gridDim.x;
    for (auto i = tid; i < element_count; i += stride) {
        data[tid].x *= scale;
        data[tid].y *= scale;
    }
}

int main() {
    cuda_stream_wrapper stream_wrapper;
    const auto &stream = stream_wrapper.get_stream();

    int dim = 1024;
    hipfftHandle plan;
    hipfftPlan1d(&plan, dim, HIPFFT_Z2Z, 1);
    hipfftSetStream(plan, stream);

    std::vector<hipDoubleComplex> h_in(dim, make_hipDoubleComplex(0, 0));
    for (int i = 3; i < dim; i++) {
        h_in[i] = make_hipDoubleComplex(1.0f / i, -2.0f / i);
    }

    for (int i = 0; i < dim; i++) {
        printf("%f + %fi, ", h_in[i].x, h_in[i].y);
    }
    printf("\n");

    auto d_data = make_cuda_auto_ptr<hipDoubleComplex>(dim, stream);
    hipMemcpyAsync(d_data.get(), h_in.data(), dim * sizeof(hipDoubleComplex), hipMemcpyHostToDevice,
                    stream);

    hipfftExecZ2Z(plan, d_data.get(), d_data.get(), HIPFFT_FORWARD);

    scaling_kernel<<<dim / 128, 128, 0, stream>>>(
            d_data.get(), dim, 1.f / dim);

    hipfftExecZ2Z(plan, d_data.get(), d_data.get(), HIPFFT_BACKWARD);

    std::vector<hipDoubleComplex> h_out(dim);
    hipMemcpyAsync(h_out.data(), d_data.get(), dim * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost,
                    stream);

    hipStreamSynchronize(stream);

    for (int i = 0; i < dim; i++) {
        printf("%f + %fi, ", h_out[i].x, h_out[i].y);
    }
    printf("\n");

    hipfftDestroy(plan);

    for (int i = 0; i < dim; i++) {
        if (!(h_in[i] == h_out[i])) {
            throw std::logic_error("Error");
        }
    }

    return 0;
}